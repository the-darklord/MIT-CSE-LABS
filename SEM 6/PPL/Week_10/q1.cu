#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void SPMV_CSR(int numRows,int* data,int* colIndex,int* rowPtr,int* x,int* y)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    if(row<numRows)
    {
        int dot = 0;
        int start = rowPtr[row];
        int end = rowPtr[row+1];
        for(int ele=start;ele<end;ele++)
        {
            dot += data[ele]*x[colIndex[ele]];
        }
        y[row] = dot;
    }
}

int main()
{
    int r1,c1,r2,c2;

    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r1,&c1);
    r2 = c1;
    c2 = 1;

    int *A = (int*)malloc(r1*c1*sizeof(int));
    int *B = (int*)malloc(r2*c2*sizeof(int));
    int *C = (int*)malloc(r1*c2*sizeof(int));

    printf("Enter Matrix A : \n");
    readMatrix(A,r1,c1);

    printf("\nEnter Matrix B : \n");
    readMatrix(B,r2,c2);

    int *data = (int*)malloc(r1*c1*sizeof(int));
    int *colIndex = (int*)malloc(r1*c1*sizeof(int));
    int *rowPtr = (int*)malloc((r1+1)*sizeof(int));

    int k=0,l=0;
    int cur = -1;

    for(int i=0;i<r1;i++)
    {
        int zeroRow = 1;
        for(int j=0;j<c1;j++)
        {
            if(A[i*c1+j]!=0)
            {
                zeroRow = 0;
                data[l] = A[i*c1+j];
                colIndex[l] = j;
                if(cur!=i)
                {
                    rowPtr[k++] = l;
                    cur = i;
                }
                l++;
            }
        }
        if(zeroRow)
        {
            rowPtr[k++] = l;
        }
    }
    rowPtr[k++] = l;

    int *dData,*dColIndex,*dRowPtr,*dB,*dC;
    hipMalloc(&dData,l*sizeof(int));
    hipMalloc(&dColIndex,l*sizeof(int));
    hipMalloc(&dRowPtr,k*sizeof(int));
    hipMalloc(&dB,r2*c2*sizeof(int));
    hipMalloc(&dC,r1*c2*sizeof(int));

    hipMemcpy(dData,data,l*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dColIndex,colIndex,l*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dRowPtr,rowPtr,k*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dB,B,r2*c2*sizeof(int),hipMemcpyHostToDevice);
    SPMV_CSR<<<1,r1>>>(r1,dData,dColIndex,dRowPtr,dB,dC);
    hipMemcpy(C,dC,r1*c2*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant Matrix : \n");
    printMatrix(C,r1,c2);

    hipFree(dData);
    hipFree(dColIndex);
    hipFree(dRowPtr);
    hipFree(dB);
    hipFree(dC);

    return 0;
}