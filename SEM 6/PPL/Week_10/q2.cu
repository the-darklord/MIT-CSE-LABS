#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void rowPower(int* A,int* B,int c)
{
    int i = threadIdx.x;
    for(int j=0;j<c;j++)
    {
        int power = 1;
        for(int k=0;k<=i;k++)
        {
            power *= A[i*c+j];
        }
        B[i*c+j] = power;
    }
}

int main()
{
    int r,c;

    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r,&c);

    int *A = (int*)malloc(r*c*sizeof(int));
    int *B = (int*)malloc(r*c*sizeof(int));

    printf("Enter Matrix A : \n");
    readMatrix(A,r,c);

    printf("Matrix A : \n");
    printMatrix(A,r,c);

    int *dA,*dB;
    hipMalloc(&dA,r*c*sizeof(int));
    hipMalloc(&dB,r*c*sizeof(int));

    hipMemcpy(dA,A,r*c*sizeof(int),hipMemcpyHostToDevice);
    rowPower<<<1,r>>>(dA,dB,c);
    hipMemcpy(B,dB,r*c*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant Matrix : \n");
    printMatrix(B,r,c);

    hipFree(dA);
    hipFree(dB);

    return 0;
}