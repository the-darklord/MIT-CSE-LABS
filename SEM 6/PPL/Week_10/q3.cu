#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void replace(int* A,int* B,int c)
{
    int i = threadIdx.x;
    int r = blockDim.x;
    for(int j=0;j<c;j++)
    {
        if(i==0||i==r-1||j==0||j==c-1)
        {
            B[i*c+j] = A[i*c+j];
        }
        else
        {
            int n = A[i*c+j];
            int a[100];
            int k=0;
            B[i*c+j] = 0;
            for(k=0;n>0;k++)    
            {
                a[k]=n%2;
                n=n/2;
            }
            for(int l=k-1;l>=0;l--)    
            {
                B[i*c+j] = B[i*c+j]*10 + 1-a[l];
            }
        }
    }
}

int main()
{
    int r,c;

    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r,&c);

    int *A = (int*)malloc(r*c*sizeof(int));
    int *B = (int*)malloc(r*c*sizeof(int));

    printf("Enter Matrix A : \n");
    readMatrix(A,r,c);

    printf("Matrix A : \n");
    printMatrix(A,r,c);

    int *dA,*dB;
    hipMalloc(&dA,r*c*sizeof(int));
    hipMalloc(&dB,r*c*sizeof(int));

    hipMemcpy(dA,A,r*c*sizeof(int),hipMemcpyHostToDevice);
    replace<<<1,r>>>(dA,dB,c);
    hipMemcpy(B,dB,r*c*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant Matrix : \n");
    printMatrix(B,r,c);

    hipFree(dA);
    hipFree(dB);

    return 0;
}