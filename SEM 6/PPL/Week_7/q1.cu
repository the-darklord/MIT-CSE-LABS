#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int *a,*b,*c;
    int *da,*db,*dc;
    int n;
    printf("Enter Array Size : \t");
    scanf("%d",&n);
    int size = n*sizeof(int);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    printf("\nEnter Array A : \n");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&a[i]);
    }
    printf("\nEnter Array B : \n");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&b[i]);
    }
    hipMalloc((void**)&da,size);
    hipMalloc((void**)&db,size);
    hipMalloc((void**)&dc,size);
    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    //Use either one
    //add<<<n,1>>>(da,db,dc); //block size as N
    //add<<<1,n>>>(da,db,dc); //N threads
    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
    printf("Result : \n");
    for(int i=0;i<n;i++)
    {
        printf("%d ",c[i]);
    }
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}