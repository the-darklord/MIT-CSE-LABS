#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void convolution(float *n,float *m,float *p,int *mWidth,int *width)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    float value = 0;
    int start = i - (*mWidth/2);
    for(int j=0;j<*mWidth;j++)
    {
        if(start+j>=0 && start+j<*width)
        {
            value += n[start+j]*m[j];
        }
    }
    p[i] = value;
}

int main(void)
{
    float *n,*m,*p;
    int width,mWidth;
    float *dn,*dm,*dp;
    int *dw,*dmw;
    printf("Enter Width and Mask Width : \n");
    scanf("%d %d",&width,&mWidth);
    int size = sizeof(float);
    n = (float*)malloc(width*size);
    m = (float*)malloc(mWidth*size);
    p = (float*)malloc(width*size);
    for(int i=0;i<width;i++)
    {
        n[i] = i+1;
    }
    for(int i=0;i<mWidth;i++)
    {
        m[i] = i+10;
    }
    hipMalloc((void**)&dn,width*size);
    hipMalloc((void**)&dm,mWidth*size);
    hipMalloc((void**)&dp,width*size);
    hipMalloc((void**)&dw,sizeof(int));
    hipMalloc((void**)&dmw,sizeof(int));
    
    hipMemcpy(dn,n,width*size,hipMemcpyHostToDevice);
    hipMemcpy(dm,m,mWidth*size,hipMemcpyHostToDevice);
    hipMemcpy(dmw,&mWidth,sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dw,&width,sizeof(int), hipMemcpyHostToDevice);
    convolution<<<1,width>>>(dn,dm,dp,dmw,dw);
    hipMemcpy(p,dp,width*size,hipMemcpyDeviceToHost);
    printf("Result : \n");
    for(int i=0;i<width;i++)
    {
        printf("%f ",p[i]);
    }
    printf("\n");
    hipFree(dn);
    hipFree(dm);
    hipFree(dp);
    hipFree(dw);
    hipFree(dmw);
    return 0;
}