#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>

__global__ void calculate(double *a,double *b)
{
    int i = threadIdx.x;
    b[i] = sin(a[i]);
}

int main()
{
    double *a,*b;
    double *da,*db;
    int n;
    printf("Enter array size : \t");
    scanf("%d",&n);
    a = (double *)malloc(n*sizeof(double));
    b = (double *)malloc(n*sizeof(double));
    for(int i=0;i<n;i++)
    {
        a[i] = i*3.14/180.0;
    }
    hipMalloc((void **)&da,n*sizeof(double));
    hipMalloc((void **)&db,n*sizeof(double));
    hipMemcpy(da,a,n*sizeof(double),hipMemcpyHostToDevice);
    calculate<<<1,n>>>(da,db);
    hipMemcpy(b,db,n*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%lf\n",b[i]);
    }
    hipFree(da);
    hipFree(db);
    return 0;
}