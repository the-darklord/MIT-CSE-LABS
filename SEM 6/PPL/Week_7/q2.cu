#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int *a,*b,*c;
    int *da,*db,*dc;
    int n = 512;
    int size = n*sizeof(int);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    for(int i=0;i<n;i++)
    {
        a[i] = i;
        b[i] = n-i;
    }
    hipMalloc((void**)&da,size);
    hipMalloc((void**)&db,size);
    hipMalloc((void**)&dc,size);
    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    add<<<2,256>>>(da,db,dc);
    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
    printf("Result : \n");
    for(int i=0;i<n;i++)
    {
        printf("%d ",c[i]);
    }
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}