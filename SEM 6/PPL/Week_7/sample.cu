#include "hip/hip_runtime.h"
#include<stdio.h>


__global__ void add(int *a,int *b,int *c)
{
    *c = *a + *b;
}

int main(void)
{
    int a,b,c;
    int *da,*db,*dc;
    int size = sizeof(int);

    hipMalloc((void**)&da,size);
    hipMalloc((void**)&db,size);
    hipMalloc((void**)&dc,size);

    a=3;
    b=5;

    hipMemcpy(da,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,&b,size,hipMemcpyHostToDevice);
    add<<<1,1>>>(da,db,dc);
    hipMemcpy(&c,dc,size,hipMemcpyDeviceToHost);
    printf("Result : %d\n",c);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}