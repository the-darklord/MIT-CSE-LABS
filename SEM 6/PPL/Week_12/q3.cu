
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_WIDTH 32
#define RADIUS 1

__global__ void convolution_1d(float *input, float *output, int width) {
    __shared__ float tile[TILE_WIDTH + 2 * RADIUS];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    tile[tx + RADIUS] = (i < width) ? input[i] : 0.0f;
    if (tx < RADIUS) {
        tile[tx] = (i - RADIUS >= 0) ? input[i - RADIUS] : 0.0f;
        tile[tx + TILE_WIDTH + RADIUS] = (i + blockDim.x < width) ? input[i + blockDim.x] : 0.0f;
    }

    __syncthreads();

    if (i < width) {
        float result = 0.0f;
        for (int j = 0; j < 2 * RADIUS + 1; j++) {
            result += tile[tx + j] * (i+1);
        }
        output[i] = result;
    }
}

int main() {
    int width = 7;
    float *input, *output;
    float *d_input, *d_output;

    input = (float*)malloc(width * sizeof(float));
    output = (float*)malloc(width * sizeof(float));

    hipMalloc(&d_input, width * sizeof(float));
    hipMalloc(&d_output, width * sizeof(float));

    for (int i = 0; i < width; i++) {
        input[i] = i+1;
    }

    hipMemcpy(d_input, input, width * sizeof(float), hipMemcpyHostToDevice);

    convolution_1d<<<(width + TILE_WIDTH - 1) / TILE_WIDTH, TILE_WIDTH>>>(d_input, d_output, width);

    hipMemcpy(output, d_output, width * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < width; i++) {
        printf("%f ", output[i]);
    }
    printf("\n");

    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}