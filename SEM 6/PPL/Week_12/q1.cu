#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void MatMulElementThreadShared(int *a,int *b,int *c)
{
    __shared__ int MDs[TILE_WIDTH][TILE_WIDTH];
    __shared__ int NDs[TILE_WIDTH][TILE_WIDTH];
    int m;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;

    int pVal = 0;
    for(m=0;m<WIDTH/TILE_WIDTH;m++)
    {
        MDs[ty][tx] = a[row*WIDTH + m*TILE_WIDTH + tx];
        NDs[ty][tx] = b[(m*TILE_WIDTH + ty)*WIDTH + col];
        __syncthreads();
        for(int k=0;k<TILE_WIDTH;k++)
        {
            pVal += MDs[ty][k]*NDs[k][tx];
        }
        __syncthreads();
    }
    c[row*WIDTH+col] = pVal;
}

int main()
{
    int *matA,*matB,*matProd;
    int *dA,*dB,*dC;

    printf("\n==Enter elements of Matrix A (4x4) ==\n");
    matA = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    for(int i=0;i<WIDTH*WIDTH;i++)
    {
        scanf("%d",&matA[i]);
    }
    printf("\n==Enter elements of Matrix B (4x4) ==\n");
    matB = (int*)malloc(sizeof(int)*WIDTH*WIDTH);
    for(int i=0;i<WIDTH*WIDTH;i++)
    {
        scanf("%d",&matB[i]);
    }
    matProd = (int*)malloc(sizeof(int)*WIDTH*WIDTH);

    hipMalloc(&dA,sizeof(int)*WIDTH*WIDTH);
    hipMalloc(&dB,sizeof(int)*WIDTH*WIDTH);
    hipMalloc(&dC,sizeof(int)*WIDTH*WIDTH);

    hipMemcpy(dA,matA,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    hipMemcpy(dB,matB,sizeof(int)*WIDTH*WIDTH,hipMemcpyHostToDevice);
    int numBlocks = WIDTH/BLOCK_WIDTH;
    dim3 grid(numBlocks,numBlocks);
    dim3 block(BLOCK_WIDTH,BLOCK_WIDTH);

    MatMulElementThreadShared<<<grid,block>>>(dA,dB,dC);

    hipMemcpy(matProd,dC,sizeof(int)*WIDTH*WIDTH,hipMemcpyDeviceToHost);

    printf("\n==Result==\n");
    printf("------------\n");
    for(int i=0;i<WIDTH;i++)
    {
        for(int j=0;j<WIDTH;j++)
        {
            printf("%6d ",matProd[i*WIDTH+j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(matA);
    free(matB);
    free(matProd);

    return 0;
}