#include "hip/hip_runtime.h"
#include<stdio.h>


__constant__ float d_N[100];
__constant__ int d_width;
__constant__ float d_M[100];
__constant__ int d_mask_width;

__global__ void convolution_1D(float *P){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float Pvalue = 0;
	int N_start_point = i - (d_mask_width/2);
	for(int j=0;j<d_mask_width;j++){
		if(N_start_point+j >=0 && N_start_point+j < d_width){
			Pvalue += d_N[N_start_point+j]*d_M[j];
		}
	}
	P[i] = Pvalue;
}

int main(void) {
	float *N, *M, *P;
	int width,mask_width;
	printf("Enter the width: ");
	scanf("%d",&width);
	printf("Enter the mask width: ");
	scanf("%d",&mask_width);
	N = (float*)malloc(width*sizeof(float));
	M = (float*)malloc(mask_width*sizeof(float));
	P = (float*)malloc(width*sizeof(float));
	float *d_c;
	int size= sizeof(float);
	for(int i=0;i<width;i++){
		N[i] = i+1;
	}
	for(int i=0;i<mask_width;i++){
		M[i] = i+10;
	}

	hipMalloc((void **)&d_c, width*size);

	hipMemcpyToSymbol(HIP_SYMBOL(d_N),N,width*size);
	hipMemcpyToSymbol(HIP_SYMBOL(d_M),M,mask_width*size);
	hipMemcpyToSymbol(HIP_SYMBOL(d_width),&width,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask_width),&mask_width,sizeof(int));

	convolution_1D<<<1,width>>>(d_c);

	hipMemcpy(P, d_c, width*size, hipMemcpyDeviceToHost);
	printf("Result:\n");
	for(int i=0;i<width;i++){
		printf("%lf ",P[i]);
	}
    printf("\n");
	hipFree(d_c);
	return 0;
}