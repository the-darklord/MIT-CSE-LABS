/*
    COPYRIGHT
*/

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void Copy(char *str, char *rstr,int* ind,int len)
{
    int i = threadIdx.x;
    for(int j=0;j<len-i;j++)
    {
        rstr[ind[i]+j] = str[j];
    }
}

int main()
{
    char str[100],rstr[300];
    printf("Enter a String : \n");
    scanf("%s", str);
    char *d_str,*d_rstr;
    int len = strlen(str);
    int* ind = (int*)malloc(len*sizeof(int));
    int *d_ind;
    ind[0] = 0;
    for(int i=1;i<len;i++)
    {
        ind[i] = ind[i-1]+len-i+1;
    }
    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_rstr, (len*len)*sizeof(char));
    hipMalloc((void**)&d_ind, len * sizeof(int));
    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_ind, ind, len * sizeof(int), hipMemcpyHostToDevice);
    Copy<<<1,len>>>(d_str,d_rstr,d_ind,len);
    hipMemcpy(rstr, d_rstr, (len*len)*sizeof(char), hipMemcpyDeviceToHost);
    printf("\nResultant String : %s\n",rstr);
    hipFree(d_str);
    hipFree(d_rstr);
    return 0;
}