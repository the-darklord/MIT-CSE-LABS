/*
    COPYRIGHT
*/

#include "hip/hip_runtime.h"


#include <stdio.h>

#define MAX_SENTENCE_LENGTH 1024
#define MAX_WORD_LENGTH 64

__global__ void countWordKernel(const char* sentence, const char* word, int* count, int sentenceLength, int wordLength) {
    int tid = threadIdx.x;

    while (tid < sentenceLength) {
        int i = 0;
        while (i < wordLength && sentence[tid + i] == word[i]) {
            i++;
        }

        if (i == wordLength) {
            atomicAdd(count, 1);
            tid += wordLength;
        } else {
            tid++;
        }
    }
}

int main() {
    char sentence[MAX_SENTENCE_LENGTH] = "This is a sample sentence. This sentence contains the word sample multiple times.";
    char word[MAX_WORD_LENGTH] = "sample";

    int sentenceLength = strlen(sentence);
    int wordLength = strlen(word);

    char* d_sentence;
    char* d_word;
    int* d_count;

    hipMalloc((void**)&d_sentence, sentenceLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sentenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_count, 0, sizeof(int));

    int blockSize = 1;
    int gridSize = 1;

    countWordKernel<<<gridSize, blockSize>>>(d_sentence, d_word, d_count, sentenceLength, wordLength);

    int result;
    hipMemcpy(&result, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word, result);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}