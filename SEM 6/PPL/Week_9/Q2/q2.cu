#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void mul(int *A,int *B,int *C,int r1,int c1)
{
    int i = threadIdx.x;
    int c2 = blockDim.x;
    for(int j=0;j<r1;j++)
    {
        int sum = 0;
        for(int k=0;k<c1;k++)
        {
            sum += A[j*c1+k]*B[k*c2+i];
        }
        C[j*c2+i] = sum;
    }
}

int main()
{
    int *A,*B,*C;
    int r1,c1,r2,c2;
    int *dA,*dB,*dC;
    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r1,&c1);
    printf("Enter Dimensions of Matrix B : \n");
    scanf("%d %d",&r2,&c2);
    if(c1!=r2)
    {
        printf("Invalid Dimensions\n");
        exit(1);
    }
    A = (int*)malloc(r1*c1*sizeof(int));
    B = (int*)malloc(r2*c2*sizeof(int));
    C = (int*)malloc(r1*c2*sizeof(int));
    printf("Enter Matrix A : \n");
    readMatrix(A,r1,c1);
    printf("Enter Matrix B : \n");
    readMatrix(B,r2,c2);
    printf("\n\n-------------------------------------------------------------------\n\n");
    printf("---------------------------Matrix A---------------------------\n");
    printMatrix(A,r1,c1);
    printf("---------------------------Matrix B---------------------------\n");
    printMatrix(B,r2,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipMalloc(&dA,r1*c1*sizeof(int));
    hipMalloc(&dB,r2*c2*sizeof(int));
    hipMalloc(&dC,r1*c2*sizeof(int));

    hipMemcpy(dA,A,r1*c1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dB,B,r2*c2*sizeof(int),hipMemcpyHostToDevice);
    mul<<<1,c2>>>(dA,dB,dC,r1,c1);
    hipMemcpy(C,dC,r1*c2*sizeof(int),hipMemcpyDeviceToHost);

    printf("---------------------------Matrix C---------------------------\n");
    printMatrix(C,r1,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}