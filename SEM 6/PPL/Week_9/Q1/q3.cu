#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void add(int *A,int *B,int *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int *A,*B,*C;
    int r1,c1,r2,c2;
    int *dA,*dB,*dC;
    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r1,&c1);
    printf("Enter Dimensions of Matrix B : \n");
    scanf("%d %d",&r2,&c2);
    if(r1!=r2 && c1!=c2)q3
    readMatrix(B,r2,c2);
    printf("\n\n-------------------------------------------------------------------\n\n");
    printf("---------------------------Matrix A---------------------------\n");
    printMatrix(A,r1,c1);
    printf("---------------------------Matrix B---------------------------\n");
    printMatrix(B,r2,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipMalloc(&dA,r1*c1*sizeof(int));
    hipMalloc(&dB,r2*c2*sizeof(int));
    hipMalloc(&dC,r1*c2*sizeof(int));

    hipMemcpy(dA,A,r1*c1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dB,B,r2*c2*sizeof(int),hipMemcpyHostToDevice);
    add<<<1,r1*c2>>>(dA,dB,dC);
    hipMemcpy(C,dC,r1*c2*sizeof(int),hipMemcpyDeviceToHost);

    printf("---------------------------Matrix C---------------------------\n");
    printMatrix(C,r1,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}