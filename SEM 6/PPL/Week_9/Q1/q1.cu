#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void readMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            scanf("%d",&M[i*c+j]);
        }
    }
}

void printMatrix(int *M,int r,int c)
{
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
            printf("%d ",M[i*c+j]);
        }
        printf("\n");
    }
}

__global__ void add(int *A,int *B,int *C,int r,int c)
{
    int i = threadIdx.x;
    for(int j=0;j<c;j++)
    {
        C[i*r+j] = A[i*r+j] + B[i*r+j];
    }
}

int main()
{
    int *A,*B,*C;
    int r1,c1,r2,c2;
    int *dA,*dB,*dC;
    printf("Enter Dimensions of Matrix A : \n");
    scanf("%d %d",&r1,&c1);
    printf("Enter Dimensions of Matrix B : \n");
    scanf("%d %d",&r2,&c2);
    if(r1!=r2 && c1!=c2)
    {
        printf("Invalid Dimensions\n");
        exit(1);
    }
    A = (int*)malloc(r1*c1*sizeof(int));
    B = (int*)malloc(r2*c2*sizeof(int));
    C = (int*)malloc(r1*c2*sizeof(int));
    printf("Enter Matrix A : \n");
    readMatrix(A,r1,c1);
    printf("Enter Matrix B : \n");
    readMatrix(B,r2,c2);
    printf("\n\n-------------------------------------------------------------------\n\n");
    printf("---------------------------Matrix A---------------------------\n");
    printMatrix(A,r1,c1);
    printf("---------------------------Matrix B---------------------------\n");
    printMatrix(B,r2,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipMalloc(&dA,r1*c1*sizeof(int));
    hipMalloc(&dB,r2*c2*sizeof(int));
    hipMalloc(&dC,r1*c2*sizeof(int));

    hipMemcpy(dA,A,r1*c1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dB,B,r2*c2*sizeof(int),hipMemcpyHostToDevice);
    add<<<1,r1>>>(dA,dB,dC,r1,c2);
    hipMemcpy(C,dC,r1*c2*sizeof(int),hipMemcpyDeviceToHost);

    printf("---------------------------Matrix C---------------------------\n");
    printMatrix(C,r1,c2);
    printf("-------------------------------------------------------------------\n\n");

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}